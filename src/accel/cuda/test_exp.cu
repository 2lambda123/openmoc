#include "hip/hip_runtime.h"
#include<iostream>
#include "dev_exponential.h"
using namespace std;

__global__ void compute_exp_on_gpu(float* x_dev)
{
  // *x_dev = 1.-__expf(*x_dev);
  *x_dev = dev_exponential(*x_dev);
}

int main()
{
  float x;
  float* dev_x;
  while (1)
  {
    cout << "enter number:\n" << endl;
    cin >> x;
    hipMalloc((void**)&dev_x, sizeof(float));
    hipMemcpy((void**)dev_x, (void**)&x, sizeof(float),
        hipMemcpyHostToDevice);
    compute_exp_on_gpu<<<1,1>>>(dev_x);
    hipMemcpy((void**)&x, (void**)dev_x, sizeof(float),
        hipMemcpyDeviceToHost);
    cout << "Device calculated: " << x << endl;
  }
}
