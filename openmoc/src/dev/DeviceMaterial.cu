#include "DeviceMaterial.h"


/**
 * @brief Given a pointer to a material on the host and a material on the 
 *        device, copy all of the properties from the material on the host 
 *        to the device.
 * @param material_h pointer to a material on the host
 * @param material_d pointer to a material on the device
 */
void cloneOnDevice(Material* material_h, dev_material* material_d) {

    /* Copy over the material's id and uid */
    int id = material_h->getId();
    int uid = material_h->getUid();
    int num_groups = material_h->getNumEnergyGroups();

    hipMemcpy((void*)&material_d->_id, (void*)&id, sizeof(int), 
	       hipMemcpyHostToDevice);
    hipMemcpy((void*)&material_d->_uid, (void*)&uid, sizeof(int), 
	       hipMemcpyHostToDevice);

    /* Allocate memory on the device for each material data array */
    double* sigma_t;
    double* sigma_a;
    double* sigma_s;
    double* sigma_f;
    double* nu_sigma_f;
    double* chi;

    /* Allocate memory on device for materials data arrays */
    hipMalloc((void**)&sigma_t, num_groups * sizeof(double));
    hipMalloc((void**)&sigma_a, num_groups * sizeof(double));
    hipMalloc((void**)&sigma_s, num_groups * num_groups * sizeof(double));
    hipMalloc((void**)&sigma_f, num_groups * sizeof(double));
    hipMalloc((void**)&nu_sigma_f, num_groups * sizeof(double));
    hipMalloc((void**)&chi, num_groups * sizeof(double));

    /* Copy materials data from host to arrays on the device */
    hipMemcpy((void*)sigma_t, (void*)material_h->getSigmaT(), 
	       num_groups * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)sigma_a, (void*)material_h->getSigmaA(), 
	       num_groups * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)sigma_s, (void*)material_h->getSigmaS(), 
	       num_groups * num_groups * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)sigma_f, (void*)material_h->getSigmaF(), 
	       num_groups * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)nu_sigma_f, (void*)material_h->getNuSigmaF(), 
	       num_groups * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)chi, (void*)material_h->getChi(), 
	       num_groups * sizeof(double), hipMemcpyHostToDevice);

    /* Copy materials data pointers to device material */
    hipMemcpy((void*)&material_d->_sigma_t, (void*)&sigma_t, sizeof(double*), 
                hipMemcpyHostToDevice);
    hipMemcpy((void*)&material_d->_sigma_a, (void*)&sigma_a, sizeof(double*), 
                hipMemcpyHostToDevice);
    hipMemcpy((void*)&material_d->_sigma_s, (void*)&sigma_s, sizeof(double*), 
                hipMemcpyHostToDevice);
    hipMemcpy((void*)&material_d->_sigma_f, (void*)&sigma_f, sizeof(double*), 
                hipMemcpyHostToDevice);
    hipMemcpy((void*)&material_d->_nu_sigma_f, (void*)&nu_sigma_f, 
                sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy((void*)&material_d->_chi, (void*)&chi, sizeof(double*), 
                hipMemcpyHostToDevice);

    return;
}
