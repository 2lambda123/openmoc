#include "hip/hip_runtime.h"
#include "DeviceSolver.h"


/** The number of azimuthal angles */
__constant__ int _num_azim_devc[1];

/** The number of energy groups */
__constant__ int _num_groups_devc[1];

/** The number of flat source regions */
__constant__ int _num_FSRs_devc[1];

/** The number of polar angles */
__constant__ int _num_polar_devc[1];

/** Twice the number of polar angles */
__constant__ int _two_times_num_polar_devc[1];

/** The number of polar angles times energy groups */
__constant__ int _polar_times_groups_devc[1];

/** An array of the weights for the polar angles from the quadrature set */
__constant__ FP_PRECISION _polar_weights_devc[MAX_POLAR_ANGLES*MAX_AZIM_ANGLES];

/** A pointer to an array with the number of tracks per azimuthal angle */
__constant__ int _num_tracks_devc[MAX_AZIM_ANGLES/2];

/** The total number of tracks */
__constant__ int _tot_num_tracks_devc[1];

/** An array of the cumulative number of tracks for each azimuthal angle */
__constant__ int _track_index_offsets_devc[MAX_AZIM_ANGLES/2];

/** The maximum index of the exponential prefactor array */
__constant__ int _prefactor_max_index_devc[1];

/** The spacing for the exponential prefactor array */
__constant__ FP_PRECISION _prefactor_spacing_devc[1];

/** The inverse spacing for the exponential prefactor array */
__constant__ FP_PRECISION _inverse_prefactor_spacing_devc[1];


/**
 * @brief Set the scalar flux for each energy group inside each 
 *        dev_flatsourceregion to a constant value.
 * @param value the value to assign to each flat source region flux
 */
__global__ void flattenFSRFluxesOnDevice(FP_PRECISION* scalar_flux, 
					 FP_PRECISION* old_scalar_flux,
					 FP_PRECISION value) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    /* Loop over all FSRs and energy groups */
    while (tid < *_num_FSRs_devc) {
        for (int e=0; e < *_num_groups_devc; e++) {
            scalar_flux(tid,e) = value;
  	    old_scalar_flux(tid,e) = value;
         }

	tid += blockDim.x * gridDim.x;
     }

    return;
}


/**
 * @brief Set the source for each energy group inside each dev_flatsourceregion
 *        to a constant value.
 * @param value the value to assign to each flat source region source
 */
__global__ void flattenFSRSourcesOnDevice(FP_PRECISION* source, 
					  FP_PRECISION* old_source,
					  FP_PRECISION value) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < *_num_FSRs_devc) {
        for (int e=0; e < *_num_groups_devc; e++) {
	    source(tid,e) = value;
	    old_source(tid,e) = value;
	}

	tid += blockDim.x * gridDim.x;
    }

    return;
}


/**
 * @brief Zero each track's boundary fluxes for each energy group and polar
 *        angle in the "forward" and "reverse" directions.
 * @param boundary_flux array of angular fluxes for each track and energy group
 */
__global__ void zeroTrackFluxesOnDevice(FP_PRECISION* boundary_flux) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    /* Loop over all tracks and energy groups and set each track's 
     * incoming and outgoing flux to zero */
    while(tid < *_tot_num_tracks_devc) {
        for (int pe2=0; pe2 < 2*(*_polar_times_groups_devc); pe2++)
    	    boundary_flux(tid,pe2) = 0.0;

	tid += blockDim.x * gridDim.x;
    }

    return;
}


/**
* Compute the total fission source from all flat source regions
* @param FSRs pointer to the flat source region array on the device
* @param num_FSRs pointer to an int of the number of flat source regions
* @param materials pointer an array of materials on the device
* @param fission_source pointer to the value for the total fission source
*/
__global__ void computeFissionSourcesOnDevice(dev_flatsourceregion* FSRs,
					      dev_material* materials,
					      FP_PRECISION* scalar_flux,
					      FP_PRECISION* fission_source) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ FP_PRECISION shared_fission_source[];
    dev_flatsourceregion* curr_FSR;
    dev_material* curr_material;
    double* nu_sigma_f;
    FP_PRECISION volume;

    /* Initialize fission source to zero */
    shared_fission_source[threadIdx.x] = 0;

    /* Iterate over all FSRs */
    while (tid < *_num_FSRs_devc) {

        curr_FSR = &FSRs[tid];
	curr_material = &materials[curr_FSR->_material_uid];
	nu_sigma_f = curr_material->_nu_sigma_f;
	volume = curr_FSR->_volume;

	/* Iterate over all energy groups and update
	 * fission source for this block */
	for (int e=0; e < *_num_groups_devc; e++)
	    shared_fission_source[threadIdx.x] += 
	        nu_sigma_f[e] * scalar_flux(tid,e) * volume;

	/* Increment thread id */
	tid += blockDim.x * gridDim.x;
    }

    /* Copy this threads fission source to global memory */
    tid = threadIdx.x + blockIdx.x * blockDim.x;
    fission_source[tid] = shared_fission_source[threadIdx.x];
    
    return;
}


/**
 * @brief Normalizes all flatsourceregion scalar fluxes and track boundary
 *        angular fluxes to the total fission source (times nu).
 */
__global__ void normalizeFluxesOnDevice(FP_PRECISION* scalar_flux, 
					FP_PRECISION* boundary_flux, 
					FP_PRECISION norm_factor) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    /* Normalize scalar fluxes for each flat source region */
    
    while(tid < *_num_FSRs_devc) {
        for (int e=0; e < *_num_groups_devc; e++)
	  scalar_flux(tid,e) *= norm_factor;

	tid += blockDim.x * gridDim.x;
    }

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    /* Normalize angular boundary fluxes for each track */
    while(tid < *_tot_num_tracks_devc) {
        for (int pe2=0; pe2 < 2*(*_polar_times_groups_devc); pe2++)
	    boundary_flux(tid,pe2) *= norm_factor;

	tid += blockDim.x * gridDim.x;
    }

    return;
}


/**
 * @brief Computes the total source (fission and scattering) in each flat 
 *        source region.
 * @details This method computes the total source in each region based on
 *          this iteration's current approximation to the scalar flux. A
 *          residual for the source with respect to the source compute on
 *          the previous iteration is computed and returned. The residual
 *          is determined as follows:
 *          /f$ res = \sqrt{\frac{\displaystyle\sum \displaystyle\sum 
 *                    \left(\frac{Q^i - Q^{i-1}{Q^i}\right)^2}{# FSRs}} \f$
 *
 * @return the residual between this source and the previous source
 */
 __global__ void computeFSRSourcesOnDevice(dev_flatsourceregion* FSRs,
					   dev_material* materials,
					   FP_PRECISION* scalar_flux,
					   FP_PRECISION* source,
					   FP_PRECISION* old_source,
					   FP_PRECISION* ratios,
					   FP_PRECISION inverse_k_eff,
					   FP_PRECISION* source_residual) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    /* Reset the residual for the old and new fission sources to zero */
    source_residual[threadIdx.x + blockIdx.x * blockDim.x] = 0.0;

    FP_PRECISION fission_source;
    FP_PRECISION scatter_source;

    dev_flatsourceregion* curr_FSR;
    dev_material* curr_material;

    double* nu_sigma_f;
    double* sigma_s;
    double* sigma_t;
    double* chi;

    /* Iterate over all FSRs */
    while (tid < *_num_FSRs_devc) {

	curr_FSR = &FSRs[tid];
	curr_material = &materials[curr_FSR->_material_uid];

	nu_sigma_f = curr_material->_nu_sigma_f;
	sigma_s = curr_material->_sigma_s;
	sigma_t = curr_material->_sigma_t;
	chi = curr_material->_chi;

	/* Initialize the fission source to zero for this region */
	fission_source = 0;
	
	/* Compute total fission source for current region */
	for (int e=0; e < *_num_groups_devc; e++)
	    fission_source += scalar_flux(tid,e) * nu_sigma_f[e];
      
	/* Compute total scattering source for region for group G */
	for (int G=0; G < *_num_groups_devc; G++) {
	    scatter_source = 0;
	
	    for (int g=0; g < *_num_groups_devc; g++)
	        scatter_source += 
		    sigma_s[G*(*_num_groups_devc)+g] * scalar_flux(tid,g);
	
	    /* Set the total source for this region in this group */
	    source(tid,G) = (inverse_k_eff * fission_source * chi[G] +
			     scatter_source) * ONE_OVER_FOUR_PI;

	    ratios(tid,G) = __fdividef(source(tid,G), sigma_t[G]);
	
	    /* Compute the norm of residuals of the sources for convergence */
	    if (fabs(source(tid,G)) > 1E-10)
	        source_residual[threadIdx.x + blockIdx.x * blockDim.x] +=
		    pow((source(tid,G) - old_source(tid,G)) /
		         source(tid,G), 2);

	    /* Update the old source */	
	    old_source(tid,G) = source(tid,G);
	}
	
	/* Increment the thread id */
	tid += blockDim.x * gridDim.x;
    }

    return;
}


/**
* Compute the total fission source from all flat source regions
* @param FSRs pointer to the flat source region array on the device
* @param num_FSRs pointer to an int of the number of flat source regions
* @param materials pointer an array of materials on the device
* @param fission_source pointer to the value for the total fission source
*/
__global__ void computeFissionAndAbsorption(dev_flatsourceregion* FSRs,
					    dev_material* materials,
					    FP_PRECISION* scalar_flux,
					    FP_PRECISION* tot_absorption,
					    FP_PRECISION* tot_fission) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    dev_flatsourceregion* curr_FSR;
    dev_material* curr_material;
    double* nu_sigma_f;
    double* sigma_a;
    FP_PRECISION volume;

    FP_PRECISION absorption;
    FP_PRECISION fission;

    /* Iterate over all FSRs */
    while (tid < *_num_FSRs_devc) {

	absorption = 0;
	fission = 0;
        
	curr_FSR = &FSRs[tid];
	curr_material = &materials[curr_FSR->_material_uid];
	nu_sigma_f = curr_material->_nu_sigma_f;
	sigma_a = curr_material->_sigma_a;
	volume = curr_FSR->_volume;

	/* Iterate over all energy groups and update
	 * fission and absorption rates for this block */
	for (int e=0; e < *_num_groups_devc; e++) {
	    absorption += sigma_a[e] * scalar_flux(tid,e) * volume;
	    fission += nu_sigma_f[e] * scalar_flux(tid,e) * volume;
	}

	/* Increment thread id */
	tid += blockDim.x * gridDim.x;
    }

    /* Copy this thread's fission and absorption rates to global memory */
    tid = threadIdx.x + blockIdx.x * blockDim.x;
    tot_absorption[tid] = absorption;
    tot_fission[tid] = fission;
    
    return;
}


/**
 * @brief Compute the index into the exponential prefactor hashtable.
 * @details This method computes the index into the exponential prefactor
 *          hashtable for a segment length multiplied by the total 
 *          cross-section of the material the segment resides in.
 * @param sigm_t_l the cross-section multiplied by segment length
 * @return the hasthable index
 */ 
__device__ int computePrefactorIndex(FP_PRECISION sigma_t_l) {
    int index = sigma_t_l * *_inverse_prefactor_spacing_devc;
    index *= *_two_times_num_polar_devc;
    return index;
}


__device__ double atomicAdd(double* address, double val) {

    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
	old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +
			__longlong_as_double(assumed)));
    } while (assumed != old);
  
    return __longlong_as_double(old);
}



/**
* This kernel integrates the neutron transport equation in the "forward"
* direction along each track in the geometry using exponential prefactors
* which are precomputed and stored in a hash table for O(1) lookup and
* interpolation
*/
__global__ void transportSweepOnDevice(FP_PRECISION* scalar_flux,
				       FP_PRECISION* boundary_flux,
				       FP_PRECISION* ratios,
				       FP_PRECISION* leakage,
				       dev_material* materials,
				       dev_track* tracks,
				       FP_PRECISION* prefactor_array,
				       int tid_offset,
				       int tid_max) {

    int tid = tid_offset + threadIdx.x + blockIdx.x * blockDim.x;

    int polar_times_groups = *_polar_times_groups_devc;
    FP_PRECISION* polar_weights = _polar_weights_devc;

    int index_offset = threadIdx.x * (*_two_times_num_polar_devc + 1);
    int energy_group = tid % (*_num_groups_devc);
    int energy_angle_index = energy_group * (*_num_polar_devc);
    int fsr_flux_index = index_offset + (*_two_times_num_polar_devc);
    int track_flux_index;

    int fsr_id;
    int track_id = int(tid / *_num_groups_devc);
    int track_out_id;
    bool bc;
    int start;
    int pe;
    int azim_angle_index;

    dev_track* curr_track;
    dev_segment* curr_segment;
    dev_material* curr_material;
    int num_segments;
    FP_PRECISION delta;
    
    double* sigma_t;

    /* temporary flux for track and fsr fluxes */
    extern __shared__ FP_PRECISION temp_flux[];

    /* Indices for exponential prefactor hashtable */
    FP_PRECISION sigma_t_l;
    int index;

    /* Iterate over track with azimuthal angles in (0, pi/2) */
    while (track_id < tid_max) {

        /* Initialize local registers with important data */
        curr_track = &tracks[track_id];
        azim_angle_index = curr_track->_azim_angle_index;
      	num_segments = curr_track->_num_segments;
      
	/* Put track's flux in the shared memory temporary flux array */
      	for (int p=0; p < *_num_polar_devc; p++) {
	
	    /* Forward flux along this track */
      	    pe = energy_angle_index + p;
	    temp_flux[index_offset + p] = boundary_flux(track_id,pe);
	
	    /* Reverse flux along this track */
      	    pe = polar_times_groups + energy_angle_index + p;
	    temp_flux[index_offset + *_num_polar_devc + p] = 
	    boundary_flux(track_id,pe);
      	}

      	track_flux_index = index_offset;
      
	/* Loop over each segment in forward direction */
	for (int i=0; i < num_segments; i++) {

	    curr_segment = &curr_track->_segments[i];
	    fsr_id = curr_segment->_region_uid;
	    curr_material = &materials[curr_segment->_material_uid];
	    sigma_t = curr_material->_sigma_t;

	    /* Zero the FSR scalar flux contribution from this segment 
	     * and energy group */
	    temp_flux[fsr_flux_index] = 0.0;

	    /* Compute the exponential prefactor hashtable index */
	    sigma_t_l = sigma_t[energy_group] * curr_segment->_length;
	    index = computePrefactorIndex(sigma_t_l);
	
	    /* Loop over polar angles */
	    for (int p=0; p < *_num_polar_devc; p++) {
	      delta = (temp_flux[track_flux_index+p] - 
		       ratios(fsr_id,energy_group)) * 
	               prefactor(index,p,sigma_t_l);
		temp_flux[fsr_flux_index] += delta * polar_weights[p];
	    	temp_flux[track_flux_index+p] -= delta;
	    }


	    /* Increment the scalar flux for this flat source region */
	    atomicAdd(&scalar_flux(fsr_id,energy_group), 
	  	      temp_flux[fsr_flux_index]);
	}
      
	/* Transfer flux to outgoing track */
	track_out_id = curr_track->_track_out;
	bc = curr_track->_bc_out;
	start = curr_track->_refl_out * polar_times_groups;

	/* Put track's flux in the shared memory temporary flux array */
      	for (int p=0; p < *_num_polar_devc; p++) {
	
	    /* Forward flux along this track */
      	    pe = energy_angle_index + p;
	    boundary_flux(track_out_id,start+pe) = 
	        temp_flux[track_flux_index+p] * bc;
	    leakage[threadIdx.x + blockIdx.x * blockDim.x] +=
	        temp_flux[track_flux_index+p] * 
	        polar_weights(azim_angle_index,pe % (*_num_polar_devc)) * (!bc);
      	}

	/* Loop over each segment in reverse direction */
	track_flux_index = index_offset + (*_num_polar_devc);

	for (int i=num_segments-1; i > -1; i--) {

	    curr_segment = &curr_track->_segments[i];
	    fsr_id = curr_segment->_region_uid;
	    curr_material = &materials[curr_segment->_material_uid];
	    sigma_t = curr_material->_sigma_t;

	    /* Zero the FSR scalar flux contribution from this segment 
	     * and energy group */
	    temp_flux[fsr_flux_index] = 0.0;

	    /* Compute the exponential prefactor hashtable index */
	    sigma_t_l = sigma_t[energy_group] * curr_segment->_length;
	    index = computePrefactorIndex(sigma_t_l);
	
	    /* Loop over polar angles */
	    for (int p=0; p < *_num_polar_devc; p++) {
	        delta = (temp_flux[track_flux_index+p] - 
	    		 ratios(fsr_id,energy_group)) * 
		         prefactor(index,p,sigma_t_l);

		//FIXME: Is this the correct way to inex into polar weights?
		temp_flux[fsr_flux_index] += delta * polar_weights[p];
	    	temp_flux[track_flux_index+p] -= delta;
	    }

	    /* Increment the scalar flux for this flat source region */
	    atomicAdd(&scalar_flux(fsr_id,energy_group), 
		      temp_flux[fsr_flux_index]);
	}
      
	/* Transfer flux to outgoing track */
	track_out_id = curr_track->_track_in;
	bc = curr_track->_bc_in;
	start = curr_track->_refl_in * polar_times_groups;

	/* Put track's flux in the shared memory temporary flux array */
      	for (int p=0; p < *_num_polar_devc; p++) {
	
	    /* Forward flux along this track */
      	    pe = energy_angle_index + p;
	    boundary_flux(track_out_id,start+pe) = 
	        temp_flux[track_flux_index+p] * bc;
	    leakage[threadIdx.x + blockIdx.x * blockDim.x] +=
	        temp_flux[track_flux_index+p] * 
	        polar_weights(azim_angle_index,pe % (*_num_polar_devc)) * (!bc);
      	}

	tid += blockDim.x * gridDim.x;
        track_id = int(tid / *_num_groups_devc);
	energy_group = tid % (*_num_groups_devc);
	energy_angle_index = energy_group * (*_num_polar_devc);
    }

    return;
}



/**
* Normalizes the flux to the volume of each FSR and adds in the source term
* computed and stored in the ratios attribute for each FSR
*/
__global__ void normalizeFluxToVolumeOnDevice(FP_PRECISION* scalar_flux,
					      FP_PRECISION* ratios,
					      dev_flatsourceregion* FSRs,
					      dev_material* materials) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    dev_flatsourceregion* curr_FSR;
    FP_PRECISION volume;
    
    dev_material* curr_material;
    double* sigma_t;

    /* Iterate over all FSRs */
    while (tid < *_num_FSRs_devc) {

        curr_FSR = &FSRs[tid];
	curr_material = &materials[curr_FSR->_material_uid];
	volume = curr_FSR->_volume;
	sigma_t = curr_material->_sigma_t;
	
	/* Iterate over all energy groups */
	for (int i=0; i < *_num_groups_devc; i++) {
	    scalar_flux(tid,i) *= 0.5;
	    scalar_flux(tid,i) = FOUR_PI * ratios(tid,i) + 
	      __fdividef(scalar_flux(tid,i), (sigma_t[i] * volume));
	}

	/* Increment thread id */
	tid += blockDim.x * gridDim.x;
    }

    return;
}




/**
 * DeviceSolver constructor
 * @param geom pointer to the geometry
 * @param track_generator pointer to the TrackGenerator on the CPU
 */
DeviceSolver::DeviceSolver(Geometry* geometry, TrackGenerator* track_generator) {

    /**************************************************************************/
    /*                        Host data initialization                        */
    /**************************************************************************/

    /* The default number of threadblocks and threads per threadblock */
    _B = 64;
    _T = 64;

    if (geometry != NULL)
        setGeometry(geometry);
    else
        _geometry = NULL;

    if (track_generator != NULL)
        setTrackGenerator(track_generator);
    else {
        _track_generator = NULL;
	_host_tracks = NULL;
	_num_tracks = NULL;
    }

    /* Default polar quadrature */
    _quad = NULL;
    _quadrature_type = TABUCHI;
    _num_polar = 3;
    _two_times_num_polar = 2 * _num_polar;

    _num_iterations = 0;
    _converged_source = false;
    _source_convergence_thresh = 1E-3;
    _flux_convergence_thresh = 1E-5;


    /**************************************************************************/
    /*                       Device data initialization                       */
    /**************************************************************************/

    _FSRs = NULL;
    _materials = NULL;
    _dev_tracks = NULL;
    _track_index_offsets = NULL;

    _boundary_flux = NULL;
    _scalar_flux = NULL;
    _old_scalar_flux = NULL;
    _source = NULL;
    _old_source = NULL;
    _ratios = NULL;

    _fission_source = NULL;
    _tot_absorption = NULL;
    _tot_fission = NULL;
    _source_residual = NULL;

    _FSRs_to_powers = NULL;
    _FSRs_to_pin_powers = NULL;

    _prefactor_array = NULL;
}



/**
 * Solver destructor frees all memory on the device
 */
DeviceSolver::~DeviceSolver() {

    log_printf(NORMAL, "Cleaning up memory on the device...");

    /* Free FSRs, materials and tracks on device */
    if (_FSRs != NULL)
        hipFree(_FSRs);

    if (_materials != NULL)
        hipFree(_materials);

    if (_dev_tracks != NULL)
        hipFree(_dev_tracks);

    if (_boundary_flux != NULL)
        hipFree(_boundary_flux);

    if (_scalar_flux != NULL)
        hipFree(_scalar_flux);

    if (_old_scalar_flux != NULL)
        hipFree(_old_scalar_flux);

    if (_source != NULL)
        hipFree(_source);

    if (_old_source != NULL)
        hipFree(_old_source);

    if (_ratios != NULL)
        hipFree(_ratios);

    if (_FSRs_to_powers != NULL)
        hipFree(_FSRs_to_powers);

    if (_FSRs_to_pin_powers != NULL)
        hipFree(_FSRs_to_pin_powers);

    if (_fission_source != NULL)
        _fission_source_vec.clear();

    if (_tot_absorption != NULL)
        _tot_absorption_vec.clear();

    if (_tot_fission != NULL)
        _tot_fission_vec.clear();

    if (_source_residual != NULL)
        _source_residual_vec.clear();

    if (_leakage != NULL)
        _leakage_vec.clear();

    if (_prefactor_array != NULL)
        hipFree(_prefactor_array);
}


/**
 * @brief Returns a pointer to the geometry for this solver.
 * @return a pointer to the geometry
 */
Geometry* DeviceSolver::getGeometry() {

    if (_geometry == NULL)
        log_printf(ERROR, "Unable to return the device solver's geometry since "
		   "it has not yet been set");

    return _geometry;
}


/**
 * @brief Returns a pointer to the geometry for this solver.
 * @return a pointer to the geometry
 */
TrackGenerator* DeviceSolver::getTrackGenerator() {

    if (_track_generator == NULL)
        log_printf(ERROR, "Unable to return the device solver's track "
		   "genetrator since it has not yet been set");

    return _track_generator;
}


/**
 * @brief Returns the number of angles used for the polar quadrature.
 * @return the number of polar angles
 */
int DeviceSolver::getNumPolarAngles() {
    return _num_polar;
}


/**
 * @brief Returns the type of polar quadrature in use (TABUCHI or LEONARD).
 * @return the type of polar quadrature
 */
quadratureType DeviceSolver::getPolarQuadratureType() {
    return _quadrature_type;
}


/**
 * @brief Returns the number of transport sweeps to converge the source.
 * @return the number of iterations
 */
int DeviceSolver::getNumIterations() {
    return _num_iterations;
}


/**
 * @brief Returns the threshold for source convergence.
 * @return the threshold for source convergence
 */
FP_PRECISION DeviceSolver::getSourceConvergenceThreshold() {
    return _source_convergence_thresh;
}


/**
 * @brief Returns the threshold for flux convergence in fixed source iteration
 *        after the source has converged.
 * @return the threshold for flux convergence
 */
FP_PRECISION DeviceSolver::getFluxConvergenceThreshold() {
    return _flux_convergence_thresh;
}


/**
 * @brief
 * @details
 */
FP_PRECISION DeviceSolver::getFSRScalarFlux(int fsr_id, int energy_group) {

    /* Error checking */
    if (fsr_id >= _num_FSRs)
        log_printf(ERROR, "Unable to return a scalar flux for FSR id = %d "
		 "in enery group %d since the solver only contains FSR with "
		   "IDs greater than or equal to %d", 
		   fsr_id, energy_group, _num_FSRs-1);

    if (fsr_id < 0)
        log_printf(ERROR, "Unable to return a scalar flux for FSR id = %d "
		  "in energy group %d since FSRs do not have negative IDs", 
		  fsr_id, energy_group);

    if (energy_group-1 >= _num_groups)
        log_printf(ERROR, "Unable to return a scalar flux for FSR id = %d "
		   "in energy group %d since the solver only has %d energy "
		   "groups", fsr_id, energy_group, _num_groups);

    if (energy_group <= 0)
        log_printf(ERROR, "Unable to return a scalar flux for FSR id = %d "
		 "in energy group %d since energy groups are greater than 1",
		 fsr_id, energy_group);

    /* Copy the scalar flux for this FSR and energy group from the device */
    FP_PRECISION fsr_scalar_flux;
    int flux_index = fsr_id * _num_groups + energy_group - 1;
    hipMemcpy((void*)&fsr_scalar_flux, (void*)&_scalar_flux[flux_index], 
	       sizeof(FP_PRECISION), hipMemcpyDeviceToHost);

    return fsr_scalar_flux;
}


/**
 * @brief Return a 2D array indexed by flatsourceregion IDs and energy groups 
 *        which contains the corresponding fluxes for each flatsourceregion.
 * @return a 2D array of dev_flatsourceregion scalar fluxes
 */
FP_PRECISION* DeviceSolver::getFSRScalarFluxes() {

    if (_scalar_flux == NULL)
        log_printf(ERROR, "Unable to returns the device solver's scalar flux "
		   "array since it has not yet been allocated in memory");

    /* Copy the scalar flux for all FSRs from the device */
    FP_PRECISION* fsr_scalar_fluxes = new FP_PRECISION[_num_FSRs * _num_groups];
    hipMemcpy((void*)fsr_scalar_fluxes, (void*)_scalar_flux,
	       _num_FSRs * _num_groups * sizeof(FP_PRECISION),
	       hipMemcpyDeviceToHost);

    return _scalar_flux;
}


/**
 * @brief Return an array indexed by flatsourceregion IDs with the
 *        corresponding flatsourceregion power.
 * @return an array of flatsourceregion powers
 */
FP_PRECISION* DeviceSolver::getFSRPowers() {
    if (_FSRs_to_powers == NULL)
        log_printf(ERROR, "Unable to returns the device solver's FSR power "
		   "array since it has not yet been allocated in memory");

    return _FSRs_to_powers;
}


/**
 * @brief Return an array indexed by flatsourceregion IDs with the
 *        corresponding pin cell power.
 * @return an array of flatsourceregion pin powers
 */
FP_PRECISION* DeviceSolver::getFSRPinPowers() {
    if (_FSRs_to_pin_powers == NULL)
        log_printf(ERROR, "Unable to returns the device solver's FSR pin power "
		   "array since it has not yet been allocated in memory");

    return _FSRs_to_pin_powers;
}


/**
 * @brief Sets the geometry for the solver.
 * @details The geometry must already have initialized flat source region maps
 *          and segmentized the trackgenerator's tracks.
 * @param geometry a pointer to a geometry
 */
void DeviceSolver::setGeometry(Geometry* geometry) {

    if (geometry->getNumFSRs() == 0)
        log_printf(ERROR, "Unable to set the Geometry for the Solver "
		 "since the Geometry has not yet initialized flat "
		 "source regions");

    if (geometry->getNumEnergyGroups() == 0)
        log_printf(ERROR, "Unable to set the Geometry for the Solver "
		 "since the Geometry does not contain any energy groups");

    if (geometry->getNumMaterials() == 0)
        log_printf(ERROR, "Unable to set the Geometry for the Solver "
		 "since the Geometry does not contain any materials");

    _geometry = geometry;
    _num_FSRs = _geometry->getNumFSRs();
    _num_groups = _geometry->getNumEnergyGroups();
    _polar_times_groups = _num_polar * _num_groups;
    _num_materials = _geometry->getNumMaterials();
}


/**
 * @brief Sets the trackgenerator with characteristic tracks for the solver.
 * @details The trackgenerator must already have generated tracks and have
 *          segmentized them using the geometry.
 * @param track_generator a pointer to a trackgenerator
 */
void DeviceSolver::setTrackGenerator(TrackGenerator* track_generator) {

    if (!track_generator->containsTracks())
        log_printf(ERROR, "Unable to set the TrackGenerator for the Solver "
		 "since the TrackGenerator has not yet generated tracks");

    _track_generator = track_generator;
    _num_azim = _track_generator->getNumAzim() / 2;
    _host_tracks = _track_generator->getTracks();
    _num_tracks = _track_generator->getNumTracksArray();
    _tot_num_tracks = _track_generator->getNumTracks();
}


/**
 * @brief Sets the type of polar angle quadrature set to use (ie, TABUCHI 
 *        or LEONARD).
 * @param type the polar angle quadrature type
 */
void DeviceSolver::setPolarQuadratureType(quadratureType quadrature_type) {
    _quadrature_type = quadrature_type;
}


/**
 * @brief Sets the number of polar angles to use (only 1, 2, or 3 currently
 *        supported).
 * @param num_polar the number of polar angles
 */
void DeviceSolver::setNumPolarAngles(int num_polar) {

    if (num_polar <= 0)
        log_printf(ERROR, "Unable to set the Solver's number of polar angles "
		   "to %d since this is a negative number", num_polar);

    if (num_polar > 3)
        log_printf(ERROR, "Unable to set the DeviceSolver's number of polar "
		   "angles to %d since this is not a supported value (only 1, "
		   "2 or 3 are currently supported)", num_polar);

    _num_polar = num_polar;
    _two_times_num_polar = 2 * _num_polar;
    _polar_times_groups = _num_polar * _num_groups;
}


/**
 * @brief Sets the threshold for source convergence (>0)
 * @param source_thresh the threshold for source convergence
 */
void DeviceSolver::setSourceConvergenceThreshold(FP_PRECISION source_thresh) {

    if (source_thresh <= 0.0)
        log_printf(ERROR, "Unable to set the source convergence threshold to "
		   "%f since the threshold must be a positive number", 
		   source_thresh);

    _source_convergence_thresh = source_thresh;
}


/**
 * @brief Sets the threshold for flux convergence (>0) in fixed source
 *        iteration after the source has converged.
 * @param source_thresh the threshold for flux convergence
 */
void DeviceSolver::setFluxConvergenceThreshold(FP_PRECISION flux_thresh) {

    if (flux_thresh <= 0.0)
        log_printf(ERROR, "Unable to set the flux convergence threshold to "
	       "%f since the threshold must be a positive number",
	       flux_thresh);

    _flux_convergence_thresh = flux_thresh;
}


/**
 * @brief Sets the number of threadblocks (>0) for device kernels
 * @param num_blocks the number of threadblocks
 */
void DeviceSolver::setNumThreadBlocks(int num_blocks) {

    if (num_blocks < 0)
        log_printf(ERROR, "Unable to set the number of threadblocks to %d since "
		   "it is a negative number", num_blocks);

    _B = num_blocks;
}


/**
 * @brief Sets the number of threads per block (>0) for device kernels
 * @param num_threads the number of threads per block
 */
void DeviceSolver::setNumThreadsPerBlock(int num_threads) {

    if (num_threads < 0)
        log_printf(ERROR, "Unable to set the number of threads per block to %d "
		   "since it is a negative number", num_threads);

    _T = num_threads;
}


/**
 * @brief Allocates and initializes all memory on the device.
 * @details Memory allocated includes data necessary for transport sweeping,
 *          including tracks, segments, flat source regions, materials, 
 *          and the polar quadrature.
 */
void DeviceSolver::allocateDeviceData() {

    log_printf(NORMAL, "Allocating memory for the device solver...");


    /**************************************************************************/
    /*                             Error checking                             */
    /**************************************************************************/

    if (_track_generator == NULL)
        log_printf(ERROR, "Unable to allocate memory on the device since "
		   "the device solver does not have a pointer to the "
		   "track generator");

    if (_geometry == NULL)
        log_printf(ERROR, "Unable to allocate memory on the device since "
		   "the device solver does not have a pointer to the geometry");


    /**************************************************************************/
    /*                     Initiailze each type of memory                     */
    /**************************************************************************/

    initializeHostMemory();
    initializeGlobalMemory();
    initializeConstantMemory();

    precomputePrefactors();

    return;
}


/**
 * @brief Allocates memory for the solver on the host.
 * @details Memory allocation includes the polar quadrature, 
 */
void DeviceSolver::initializeHostMemory() {

    log_printf(INFO, "Initializing host memory for the device solver...");

    /* Initialize the a polar quadrature object on the host */
    initializePolarQuadrature();

    /* Initialize arrays of FSR powers and pin powers */
    initializePowerArrays();
}


/**
 * @brief Creates a polar quadrature object for the solver.
 */
void DeviceSolver::initializePolarQuadrature() {

    /* Deletes the old quadrature if one existed */
    if (_quad != NULL)
        delete _quad;

    _quad = new Quadrature(_quadrature_type, _num_polar);
    _polar_times_groups = _num_groups * _num_polar;
}


/**
 * @brief Allocates memory for flatsourceregion power arrays.
 * @details Deletes memory for power arrays if they were allocated from
 *          previous simulation.
 */
void DeviceSolver::initializePowerArrays() {

    /* Delete old power arrays if they exist */
    if (_FSRs_to_powers != NULL)
        delete [] _FSRs_to_powers;
    if (_FSRs_to_pin_powers != NULL)
        delete [] _FSRs_to_pin_powers;

    /* Allocate memory for FSR power and pin power arrays */
    try{
	_FSRs_to_powers = new FP_PRECISION[_num_FSRs];
	_FSRs_to_pin_powers = new FP_PRECISION[_num_FSRs];
    }
    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the device solver's "
		   "FSR power arrays. Backtrace:%s", e.what());
    }
}


/**
 * @brief
 * @details
 */
void DeviceSolver::initializeGlobalMemory() {

    log_printf(INFO, "Initializing global memory on the device...");

    initializeFSRs();
    initializeMaterials();
    initializeTracks();
    initializeFluxArrays();
    initializeSourceArrays();
    initializePowerArrays();
    initializeThrustVectors();
}


/**
 * This is a helper method for the allocateDeviceMemory method. It
 * initializes an array of dev_flatsourceregion structs on the host
 * with the appropriate values (volume, region uid, and material uid)
 * so that allocateDeviceMemory can copy the array in full to the device.
 */
void DeviceSolver::initializeFSRs() {

    log_printf(INFO, "Initializing FSRs on the device...");

    /* Delete old FSRs array if it exists */
    if (_FSRs != NULL)
        hipFree(_FSRs);

    /* Allocate memory for all tracks and track offset indices on the device */
    try{

        /* Allocate memory on device for FSRs */
        hipMalloc((void**)&_FSRs, _num_FSRs * sizeof(dev_flatsourceregion));

	/* Create a temporary FSR array to populate and then copy to device */
	dev_flatsourceregion* temp_FSRs = new dev_flatsourceregion[_num_FSRs];

	/* Get the array indexed by FSR IDs with material ID values */
	int* FSRs_to_materials = _geometry->getFSRtoMaterialMap();

	/* Iterate over all FSRs and set the UIDs and material IDs */
	for (int r=0; r < _num_FSRs; r++) {
	    temp_FSRs[r]._uid = r;
	    temp_FSRs[r]._material_uid = FSRs_to_materials[r];
	}

	/* Initialize each FSRs volume to 0 to avoid NaNs */
	for (int r=0; r < _num_FSRs; r++)
	    temp_FSRs[r]._volume = 0.0;

	Track* track;
	segment* seg;
	dev_flatsourceregion* fsr;

	double* azim_weights = _track_generator->getAzimWeights();


	/* Set each FSR's volume by accumulating the total length of all
	   tracks inside the FSR. Iterate over azimuthal angle, track, segment */
	for (int i=0; i < _num_azim; i++) {
	    for (int j=0; j < _num_tracks[i]; j++) {
	        track = &_track_generator->getTracks()[i][j];

		/* Iterate over the track's segments to update FSR volumes */
		for (int s = 0; s < track->getNumSegments(); s++) {
		    seg = track->getSegment(s);
		    fsr = &temp_FSRs[seg->_region_id];
		    fsr->_volume += seg->_length * azim_weights[i];
		}
	    }
	}

	/* Copy the temporary array of FSRs to the device */
	hipMemcpy((void*)_FSRs, (void*)temp_FSRs, 
		   _num_FSRs * sizeof(dev_flatsourceregion), 
		   hipMemcpyHostToDevice);

	/* Free the temporary array of FSRs on the host */
	free(temp_FSRs);
    }
    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the solver's flat "
		   "source regions on the device. Backtrace:%s", e.what());
    }
}


/**
 * @brief
 * @details
 */
void DeviceSolver::initializeMaterials() {

    log_printf(INFO, "Initializing materials on the device...");

    /* Delete old materials array if it exists */
    if (_materials != NULL)
        hipFree(_materials);

    /* Allocate memory for all tracks and track offset indices on the device */
    try{

	std::map<short int, Material*> host_materials=_geometry->getMaterials();
	std::map<short int, Material*>::iterator iter;

        /* Iterate through all materials and clone them on the device */
        hipMalloc((void**)&_materials, _num_materials * sizeof(dev_material));
	for (iter = host_materials.begin(); iter != host_materials.end(); ++iter)
	    cloneOnDevice(iter->second, &_materials[iter->second->getUid()]);
    }
    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the device solver's "
		   "materials. Backtrace:%s", e.what());
    }
}


/**
 * @brief
 * @details
 */
void DeviceSolver::initializeTracks() {

    log_printf(INFO, "Initializing tracks on the device...");

    /* Delete old tracks array if it exists */
    if (_dev_tracks != NULL)
        hipFree(_dev_tracks);

    /* Delete old track index offsets array if it exists */
    if (_track_index_offsets != NULL)
        delete [] _track_index_offsets;

    /* Allocate array of tracks */
    hipMalloc((void**)&_dev_tracks, _tot_num_tracks * sizeof(dev_track));

    _track_index_offsets = new int[_num_azim+1];

    /* Allocate memory for all tracks and track offset indices on the device */
    try{

        /* Iterate through all tracks and clone them on the device */
        int counter = 0;
	int index;
	for (int i=0; i < _num_azim; i++) {

            _track_index_offsets[i] = counter;

	    for (int j=0; j < _num_tracks[i]; j++) {

	        /* Clone this track on the device */
	        cloneTrack(&_host_tracks[i][j], &_dev_tracks[counter]);

		/* Make track reflective */
		index = computeScalarTrackIndex(_host_tracks[i][j].getTrackInI(),
					       _host_tracks[i][j].getTrackInJ());
		hipMemcpy((void*)&_dev_tracks[counter]._track_in,
			   (void*)&index, sizeof(int), hipMemcpyHostToDevice);

		index = computeScalarTrackIndex(_host_tracks[i][j].getTrackOutI(), 
						_host_tracks[i][j].getTrackOutJ());
		hipMemcpy((void*)&_dev_tracks[counter]._track_out, 
			   (void*)&index, sizeof(int), hipMemcpyHostToDevice);

		counter++;
	    }
	}

	_track_index_offsets[_num_azim] = counter;

    }

    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the solver's tracks "
		   "on the device. Backtrace:%s", e.what());
    }
}


/**
 * @brief Allocates memory for track boundary angular fluxes and 
 *        flatsourceregion scalar fluxes on the device.
 * @details Deletes memory for old flux arrays if they were allocated from
 *          previous simulation.
 */
void DeviceSolver::initializeFluxArrays() {

    log_printf(INFO, "Initializing flux arrays on the device...");

    /* Delete old flux arrays if they exist */
    if (_boundary_flux != NULL)
        hipFree(_boundary_flux);
    if (_scalar_flux != NULL)
        hipFree(_scalar_flux);
    if (_old_scalar_flux != NULL)
        hipFree(_old_scalar_flux);

    /* Allocate memory for all flux arrays on the device */
    try{
        hipMalloc((void**)&_boundary_flux,
		   2*_tot_num_tracks * _polar_times_groups*sizeof(FP_PRECISION));
        hipMalloc((void**)&_scalar_flux, 
		   _num_FSRs * _num_groups * sizeof(FP_PRECISION));
        hipMalloc((void**)&_old_scalar_flux, 
		   _num_FSRs * _num_groups * sizeof(FP_PRECISION));
    }
    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the solver's fluxes "
		   "on the device. Backtrace:%s", e.what());
    }
}


/**
 * @brief Allocates memory for flatsourceregion source arrays on the device.
 * @details Deletes memory for old source arrays if they were allocated from
 *          previous simulation.
 */
void DeviceSolver::initializeSourceArrays() {

    log_printf(INFO, "Initializing source arrays on the device...");

    /* Delete old sources arrays if they exist */
    if (_source != NULL)
        hipFree(_source);
    if (_old_source != NULL)
        hipFree(_old_source);
    if (_ratios != NULL)
        hipFree(_ratios);

    /* Allocate memory for all source arrays on the device */
    try{

        hipMalloc((void**)&_source, 
		   _num_FSRs * _num_groups * sizeof(FP_PRECISION));
	hipMalloc((void**)&_old_source,
		   _num_FSRs * _num_groups * sizeof(FP_PRECISION));
	hipMalloc((void**)&_ratios,
		   _num_FSRs * _num_groups * sizeof(FP_PRECISION));
    }
    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the solver's flat "
		   "source region sources array on the device. "
		   "Backtrace:%s", e.what());
    }
}


/**
 * @brief
 * @details
 */
void DeviceSolver::initializeThrustVectors() {

    log_printf(INFO, "Initializing thrust vectors on the device...");

    /* Delete old vectors if they exist */
    if (_fission_source != NULL) {
        _fission_source = NULL;
        _fission_source_vec.clear();
    }
    if (_tot_absorption != NULL) {
        _tot_absorption = NULL;
        _tot_absorption_vec.clear();
    }
    if (_tot_fission != NULL) {
        _tot_fission = NULL;
        _tot_fission_vec.clear();
    }
    if (_source_residual != NULL) {
        _source_residual = NULL;
        _source_residual_vec.clear();
    }
    if (_leakage != NULL) {
        _leakage = NULL;
        _leakage_vec.clear();
    }


    /* Allocate memory for fission, absorption and source vectors on device */
    try{
        /* Allocate fission source array on device */
        _fission_source_vec.resize(_B * _T);
	_fission_source = thrust::raw_pointer_cast(&_fission_source_vec[0]);
      
	/* Allocate total absorption reaction rate array on device */
	_tot_absorption_vec.resize(_B * _T);
	_tot_absorption = thrust::raw_pointer_cast(&_tot_absorption_vec[0]);

	/* Allocate fission reaction rate array on device */
	_tot_fission_vec.resize(_B * _T);
	_tot_fission = thrust::raw_pointer_cast(&_tot_fission_vec[0]);

	/* Allocate source residual array on device */
	_source_residual_vec.resize(_B * _T);
	_source_residual = thrust::raw_pointer_cast(&_source_residual_vec[0]);

	/* Allocate leakage array on device */
	_leakage_vec.resize(_B * _T);
	_leakage = thrust::raw_pointer_cast(&_leakage_vec[0]);
    }
    catch(std::exception &e) {
        log_printf(ERROR, "Could not allocate memory for the solver's "
		   "Thrust vectors.Backtrace:%s", e.what());
    }
}


/**
 * @brief Initializes data in constant memory on the device.
 * @details
 */
void DeviceSolver::initializeConstantMemory() {

    log_printf(INFO, "Initializing constant memory on the device...");

    /* Number of azimuthal angles */
    hipMemcpyToSymbol(HIP_SYMBOL(_num_azim_devc), (void*)&_num_azim, sizeof(int), 0, 
		       hipMemcpyHostToDevice);

    /* Number of energy groups */
    hipMemcpyToSymbol(HIP_SYMBOL(_num_groups_devc), (void*)&_num_groups, sizeof(int), 0,
		       hipMemcpyHostToDevice);

    /* Number of flat source regions */
    hipMemcpyToSymbol(HIP_SYMBOL(_num_FSRs_devc), (void*)&_num_FSRs, sizeof(int), 0,
		       hipMemcpyHostToDevice);

    /* Number of polar angles */
    hipMemcpyToSymbol(HIP_SYMBOL(_num_polar_devc), (void*)&_num_polar, sizeof(int), 0,
		       hipMemcpyHostToDevice);

    /* Twice the number of polar angles */
    hipMemcpyToSymbol(HIP_SYMBOL(_two_times_num_polar_devc), (void*)&_two_times_num_polar, 
		       sizeof(int), 0, hipMemcpyHostToDevice);

    /* Number of polar angles times energy groups */
    hipMemcpyToSymbol(HIP_SYMBOL(_polar_times_groups_devc), (void*)&_polar_times_groups, 
		       sizeof(int), 0, hipMemcpyHostToDevice);

    /* Compute polar times azimuthal angle weights */
    FP_PRECISION* polar_weights =
        (FP_PRECISION*)malloc(_num_polar * _num_azim * sizeof(FP_PRECISION));
    FP_PRECISION* multiples = _quad->getMultiples();
    double* azim_weights = _track_generator->getAzimWeights();

    for (int i=0; i < _num_azim; i++) {
        for (int j=0; j < _num_polar; j++)
	    polar_weights[i*_num_polar+j] = azim_weights[i]*multiples[j]*FOUR_PI;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(_polar_weights_devc), (void*)polar_weights,
		       _num_polar * _num_azim * sizeof(FP_PRECISION),
		       0, hipMemcpyHostToDevice);
    free(polar_weights);

    /* Array of number of tracks for each azimuthal angles */
    hipMemcpyToSymbol(HIP_SYMBOL(_num_tracks_devc), (void*)_num_tracks, 
		       _num_azim * sizeof(int), 0, hipMemcpyHostToDevice);
    
    /* Total number of tracks */
    hipMemcpyToSymbol(HIP_SYMBOL(_tot_num_tracks_devc), (void*)&_tot_num_tracks,
		       sizeof(int), 0, hipMemcpyHostToDevice);

    /* Copy the cumulative index offset for the current azimuthal angle */
    hipMemcpyToSymbol(HIP_SYMBOL(_track_index_offsets_devc), 
		       (void*)_track_index_offsets, 
		       (_num_azim+1) * sizeof(int), 0, hipMemcpyHostToDevice);
}


/**
 * @brief This method computes the index for the jth track at azimuthal angle i.
 * @details This method is necessary since the array of tracks on the device 
 *          is a 1D array which needs a one-to-one mapping from the 2D jagged 
 *          array of tracks on the host.
 * @param i azimuthal angle number
 * @param j the jth track at angle i
 * @return an index into the device track array
 */
int DeviceSolver::computeScalarTrackIndex(int i, int j) {

    int index =0;
    int p = 0;

    /* Iterate over each azimuthal angle and increment index by the number of
       tracks at each angle */
    while (p < i) {
        index += _num_tracks[p];
	p++;
    }

    /* Update index for this track since it is the jth track at angle i */
    index += j;
    
    return index;
}


/**
 * @brief Pre-computes exponential pre-factors for each segment of each track 
 *        for each polar angle and copies the table to the device. 
 * @details This method will generate a hashmap which contains values of the 
 *          pre-factor for specific segment lengths (the keys into the hashmap).
 */
void DeviceSolver::precomputePrefactors(){

    log_printf(INFO, "Building exponential prefactor hashtable on device...");

    /* Set size of prefactor array */
    int num_array_values = 10 * sqrt(1. / (8. * _source_convergence_thresh));
    FP_PRECISION prefactor_spacing = 10. / num_array_values;
    FP_PRECISION inverse_prefactor_spacing = 1.0 / prefactor_spacing;
    int prefactor_array_size = _two_times_num_polar * num_array_values;
    int prefactor_max_index = prefactor_array_size - _two_times_num_polar - 1;
    
    /* allocate arrays */
    FP_PRECISION* prefactor_array = new FP_PRECISION[prefactor_array_size];
    
    FP_PRECISION expon;
    FP_PRECISION intercept;
    FP_PRECISION slope;


    /* Create prefactor array */
    for (int i = 0; i < num_array_values; i ++){
        for (int p = 0; p < _num_polar; p++){
	    expon = exp(- (i * prefactor_spacing) / _quad->getSinTheta(p));
	    slope = - expon / _quad->getSinTheta(p);
	    intercept = expon * (1 + (i * prefactor_spacing) /
				 _quad->getSinTheta(p));
	    prefactor_array[_two_times_num_polar * i + 2 * p] = slope;
	    prefactor_array[_two_times_num_polar * i + 2 * p + 1] = intercept;
	}
    }

    /* Allocate memory for the prefactor array on the device */
    hipMalloc((void**)&_prefactor_array, 
	       prefactor_array_size * sizeof(FP_PRECISION));

    /* Copy prefactor array to the device */
    hipMemcpy((void*)_prefactor_array, (void*)prefactor_array, 
	       prefactor_array_size * sizeof(FP_PRECISION),
	       hipMemcpyHostToDevice);

    /* Copy prefactor array size and spacing to constant memory on the device */
    hipMemcpyToSymbol(HIP_SYMBOL(_prefactor_spacing_devc), (void*)&prefactor_spacing, 
		       sizeof(FP_PRECISION), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(_inverse_prefactor_spacing_devc), 
		       (void*)&inverse_prefactor_spacing, 
		       sizeof(FP_PRECISION), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(_prefactor_max_index_devc), (void*)&prefactor_max_index,
		       sizeof(int), 0, hipMemcpyHostToDevice);

    free(prefactor_array);

    return;
}


/**
 * @brief Checks that each flat source region has at least one segment within 
 *        it and if not, throw an exception and prints an error message.
 */
void DeviceSolver::checkTrackSpacing() {

    log_printf(INFO, "Checking track spacing...");

    int* FSR_segment_tallies = new int[_num_FSRs];
    std::vector<segment*> segments;
    std::vector<segment*>::iterator iter;
    Cell* cell;

    /* Set each tally to zero to begin with */
    for (int r=0; r < _num_FSRs; r++)
        FSR_segment_tallies[r] = 0;

    /* Iterate over all azimuthal angles, all tracks, and all segments
     * and tally each segment in the corresponding FSR */
    for (int i=0; i < _num_azim; i++) {
        for (int j=0; j < _num_tracks[i]; j++) {
	    segments = _host_tracks[i][j].getSegments();

            for (iter=segments.begin(); iter != segments.end(); ++iter)
	        FSR_segment_tallies[(*iter)->_region_id]++;
	}
    }

    /* Loop over all FSRs and if one FSR does not have tracks in it, print
     * error message to the screen and exit program */
    for (int r=0; r < _num_FSRs; r++) {
        if (FSR_segment_tallies[r] == 0) {
	    cell = _geometry->findCellContainingFSR(r);
	    log_printf(ERROR, "No tracks were tallied inside FSR id = %d which "
		       "is cell id = %d. Please reduce your track spacing,"
		       " increase the number of azimuthal angles, or increase "
		       "the size of the flat source regions", r, cell->getId());
	}
    }

    delete [] FSR_segment_tallies;
}


void DeviceSolver::normalizeFluxes() {

    int shared_mem = sizeof(FP_PRECISION) * _T;

    computeFissionSourcesOnDevice<<<_B, _T, shared_mem>>>(_FSRs, _materials, 
							  _scalar_flux, 
							  _fission_source);

    FP_PRECISION norm_factor = 1.0 / thrust::reduce(_fission_source_vec.begin(),
						    _fission_source_vec.end());

    normalizeFluxesOnDevice<<<_B, _T>>>(_scalar_flux, _boundary_flux, 
					norm_factor);
}


FP_PRECISION DeviceSolver::computeFSRSources() {

    computeFSRSourcesOnDevice<<<_B, _T>>>(_FSRs, _materials, _scalar_flux,
					  _source, _old_source, _ratios,
					  1.0 / _k_eff, _source_residual);

    FP_PRECISION residual = thrust::reduce(_source_residual_vec.begin(), 
					   _source_residual_vec.end());
    residual = sqrt(residual / _num_FSRs);

    return residual;
}


void DeviceSolver::transportSweep(int max_iterations) {

    int shared_mem = sizeof(FP_PRECISION) * _T * (2*_num_polar + 1);
    int tid_offset, tid_max;

    log_printf(DEBUG, "Transport sweep on device with max_iterations = %d "
	       " and # blocks = %d, # threads = %d", 
	       max_iterations, _B, _T);

    /* Loop for until converged or max_iterations is reached */
    for (int i=0; i < max_iterations; i++) {

        /* Initialize leakage to zero */
        thrust::fill(_leakage_vec.begin(), _leakage_vec.end(), 0.0);

        /* Initialize flux in each region to zero */
	tid_offset = 0;
	tid_max = _track_index_offsets[_num_azim / 2];

        flattenFSRFluxesOnDevice<<<_B, _T>>>(_scalar_flux, 
					     _old_scalar_flux, 0.0);

	transportSweepOnDevice<<<_B, _T, shared_mem>>>(_scalar_flux, 
						       _boundary_flux,
						       _ratios, _leakage,
						       _materials, _dev_tracks,
						       _prefactor_array, 
						       tid_offset, tid_max);

	tid_offset = _track_index_offsets[_num_azim / 2] * _num_groups;
	tid_max = _track_index_offsets[_num_azim];

        transportSweepOnDevice<<<_B, _T, shared_mem>>>(_scalar_flux,
						       _boundary_flux,
						       _ratios, _leakage,
						       _materials, _dev_tracks,
						       _prefactor_array,
						       tid_offset, tid_max);

	/* Add in source term, normalize fluxes to volume and save old flux */
	normalizeFluxToVolumeOnDevice<<<_B, _T>>>(_scalar_flux, _ratios, 
						  _FSRs, _materials);
    }
}


void DeviceSolver::computeKeff() {

    FP_PRECISION tot_absorption;
    FP_PRECISION tot_fission;
    FP_PRECISION tot_leakage;

    /* Compute the total fission and absorption rates on the device.
     * This kernel stores partial rates in a Thrust vector with as many
     * entries as GPU threads executed by the kernel */
    computeFissionAndAbsorption<<<_B, _T>>>(_FSRs, _materials, _scalar_flux,
					    _tot_absorption, _tot_fission);

    /* Compute the total absorption rate by reducing the partial absorption
     * rates compiled in the Thrust vector */
    tot_absorption = thrust::reduce(_tot_absorption_vec.begin(),
				    _tot_absorption_vec.end());

    /* Compute the total fission rate by reducing the partial fission
     * rates compiled in the Thrust vector */
    tot_fission = thrust::reduce(_tot_fission_vec.begin(),
				 _tot_fission_vec.end());

    /* Compute the total leakage by reducing the partial leakage
     * rates compiled in the Thrust vector */
    tot_leakage = thrust::reduce(_leakage_vec.begin(),
				 _leakage_vec.end());


    /* Compute the new keff from the fission and absorption rates */
    _k_eff = tot_fission / (tot_absorption + tot_leakage);
}


FP_PRECISION DeviceSolver::convergeSource(int max_iterations, int B, int T){
  
    /* Error checking */
    if (_geometry == NULL)
        log_printf(ERROR, "The DeviceSolver is unable to converge the source "
		   "since it does not contain a Geometry");
    if (_track_generator == NULL)
        log_printf(ERROR, "The DeviceSolver is unable to converge the source "
		   "since it does not contain a TrackGenerator");

    FP_PRECISION residual = 0.0;

    setNumThreadBlocks(B);
    setNumThreadsPerBlock(T);

    /* Initialize data structures on the device */
    allocateDeviceData();

    /* Counter for the number of iterations to converge the source */
    _num_iterations = 0;

    /* An initial guess for the eigenvalue */
    _k_eff = 1.0;

    /* Check that each FSR has at least one segment crossing it */
    checkTrackSpacing();

    /* Set scalar flux to unity for each region */
    flattenFSRFluxesOnDevice<<<_B, _T>>>(_scalar_flux, _old_scalar_flux, 1.0);
    flattenFSRSourcesOnDevice<<<_B, _T>>>(_source, _old_source, 1.0);
    zeroTrackFluxesOnDevice<<<_B, _T>>>(_boundary_flux);

    log_printf(NORMAL, "Converging the source on the device...");

    /* Source iteration loop */
    for (int i=0; i < max_iterations; i++) {

        log_printf(NORMAL, "Iteration %d on device: \tk_eff = %1.6f"
		 "\tres = %1.3E", i, _k_eff, residual);

	normalizeFluxes();
	residual = computeFSRSources();
	transportSweep(1);
	computeKeff();
	_num_iterations++;

	if (i > 1 && residual < _source_convergence_thresh){
	  //	    transportSweep(1000);
	    return _k_eff;
	}
    }

    log_printf(WARNING, "Unable to converge the source after %d iterations",
	       max_iterations);

    return _k_eff;
}
